#include "hip/hip_runtime.h"
/*
This software is part of GPU Ocean. 

Copyright (C) 2018 - 2023 SINTEF Digital
Copyright (C) 2018 - 2023 Norwegian Meteorological Institute

This CUDA kernel implements the CDKLM numerical scheme
for the shallow water equations, described in
A. Chertock, M. Dudzinski, A. Kurganov & M. Lukacova-Medvidova
Well-Balanced Schemes for the Shallow Water Equations with Coriolis Forces,
Numerische Mathematik, 138:939–973, 2017

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "common.cu"
#include "external_forcing.cu"
#include "interpolation.cu"

texture<float, hipTextureType2D> angle_tex;


//WARNING: Must match max_dt.cu and initBm_kernel.cu
//WARNING: This is error prone - as comparison with floating point numbers is not accurate
#define CDKLM_DRY_FLAG 1.0e-30f
#define CDKLM_DRY_EPS 1.0e-3f



/**
  * Returns the coriolis parameter f from the coriolis data array. 
  * @param coriolis_f_arr Array of coriolis force values to interpolate from
  * @param i Cell number along x-axis, starting from (0, 0) corresponding to first cell in domain after global ghost cells
  * @param j Cell number along y-axis
  * @param data_nx Number of cells along x axis for the coriolis array
  * @param data_ny Number of cells along y axis for the coriolis array
  * The texture is assumed to also cover the ghost cells (same shape/extent as eta)
  */
__device__
inline float coriolisF(const float* coriolis_f_arr, const int i, const int j, int data_nx, int data_ny) {
    //nx+4 to account for ghost cells
    //+0.5f to go to center of texel
    const float s = (i+0.5f) / (NX+4.0f); 
    const float t = (j+0.5f) / (NY+4.0f);
    //FIXME: Should implement so that subsampling does not get border issues, see
    //https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#table-lookup
    return bilinear_interpolation(coriolis_f_arr, data_nx, data_ny, s, t);
}



/**
  * Decompose the north vector to x and y coordinates
  * @param angle_arr Array of angle values to interpolate from
  * @param i Cell number along x-axis, starting from (0, 0) corresponding to first cell in domain after global ghost cells
  * @param j Cell number along y-axis
  * @param data_nx Number of cells along x axis for the angle array
  * @param data_ny Number of cells along y axis for the angle array
  */
__device__
inline float2 getNorth(const float* angle_arr, const int i, const int j, int data_nx, int data_ny) {
    //nx+4 to account for ghost cells
    //+0.5f to go to center of texel
    const float s = (i+0.5f) / (NX+4.0f);
    const float t = (j+0.5f) / (NY+4.0f);
    const float angle = bilinear_interpolation(angle_arr, data_nx, data_ny, s, t);
    //FIXME: Should implement so that subsampling does not get border issues, see
    //https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#table-lookup
    return make_float2(__sinf(angle), __cosf(angle));
}

// Q =[eta, u, v]
__device__ float3 CDKLM16_F_func(const float3 Q, const float H) {
    float3 F;
    const float h = Q.x + H;

    F.x = h*Q.y;                        //h*u
    F.y = h*Q.y*Q.y + 0.5f*GRAV*h*h;   //h*u*u + 0.5f*g*h*h;
    F.z = h*Q.y*Q.z;                    //h*u*v;

    return F;
}







/**
  * Note that the input vectors are (eta, u, v), thus not the regular
  * (h, hu, hv). 
  * Note also that u and v are desingularized from the start.
  */
__device__ float3 CDKLM16_flux(const float3 Qm, const float3 Qp, const float H_face) {
    
    const float hp = Qp.x + H_face;
    const float hm = Qm.x + H_face;

    // Contribution from plus cell
    float3 Fp = make_float3(0.0f, 0.0f, 0.0f);
    float up = 0.0f;
    float cp = 0.0f;
    
    if (hp > KPSIMULATOR_DEPTH_CUTOFF) {
        Fp = CDKLM16_F_func(Qp, H_face);
        up = Qp.y;         // u
        cp = sqrtf(GRAV*hp); // sqrt(GRAV*h)
    }

    // Contribution from plus cell
    float3 Fm = make_float3(0.0f, 0.0f, 0.0f);
    float um = 0.0f;
    float cm = 0.0f;

    if (hm > KPSIMULATOR_DEPTH_CUTOFF) {
        Fm = CDKLM16_F_func(Qm, H_face);
        um = Qm.y;         // u
        cm = sqrtf(GRAV*hm); // sqrt(GRAV*h)
    }
    
    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed

    // If symmetric Rieman fan, return zero flux
    if ( fabsf(ap - am) < KPSIMULATOR_FLUX_SLOPE_EPS ) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
    
    float3 F;
  
    // Q = [eta, u, v]
    // F = [hu, h*u*u + 0.5*g*h*h, h*u*v]
    F.x = ((ap*Fm.x - am*Fp.x) + ap*am*(Qp.x-Qm.x))/(ap-am);
    F.y = ((ap*Fm.y - am*Fp.y) + ap*am*(Fp.x-Fm.x))/(ap-am);

    // Balance the contribution between standard upwind and central upwind fluxes    
    F.z = (Qm.y > - Qp.y) ? FLUX_BALANCER*Fm.z : FLUX_BALANCER*Fp.z;
    F.z += (1.0f - FLUX_BALANCER)*(((ap*Fm.z - am*Fp.z) + ap*am*(hp*Qp.z - hm*Qm.z))/(ap-am));

    return F;
}



/**
  * Adjusting the slope of K_x, found in Qx[3], to avoid negative values for h on the faces,
  * in the case of dry cells
  */
__device__
void adjustSlopes_x(const int bx, const int by, 
                    float R[3][block_height+4][block_width+4],
                    float Qx[3][block_height+2][block_width+2], // used as if Qx[3][block_height][block_width + 2]
                    float Hi[block_height+3][block_width+3],
                    const int& bc_east_, const int& bc_west_,
                    const float* coriolis_f_arr) {
    
    // Need K_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), v (R[2]), H (Hi), g, dx

    
    const int j = threadIdx.y; // values in Qx
    const int l = j + 2; // values in R
    const int H_j = j + 1; // values in Hi
    
    for (int i=threadIdx.x; i<block_width+2; i+=blockDim.x) {
        // i referes to values in Qx
        const int k = i + 1; // values in R
        const int H_i = i; // values in Hi

        // Reconstruct h at east and west faces
        const float eta = R[0][l][k];
        
        float v   = R[2][l][k];
        // Fix west boundary for reconstruction of eta (corresponding to Kx)
        if ((bc_west_ == 1) && (bx + k < 2    )) { v = -v; }
        // Fix east boundary for reconstruction of eta (corresponding to Kx)
        if ((bc_east_ == 1) && (bx + k > NX+2)) { v = -v; }
        
        // Coriolis in this cell
        const float coriolis_f = coriolisF(coriolis_f_arr, bx+k, by+l, CORIOLIS_F_NX, CORIOLIS_F_NY);
        
        const float dxfv = DX*coriolis_f*v;
        
        const float H_west = 0.5f*(Hi[H_j][H_i  ] + Hi[H_j+1][H_i  ]);
        const float H_east = 0.5f*(Hi[H_j][H_i+1] + Hi[H_j+1][H_i+1]);
        
        const float h_west = eta + H_west - (Qx[2][j][i] + dxfv)/(2.0f*GRAV);
        const float h_east = eta + H_east + (Qx[2][j][i] + dxfv)/(2.0f*GRAV);
        
        // Adjust if negative water level
        Qx[2][j][i] = (h_west > 0) ? Qx[2][j][i] : -dxfv + 2.0f*GRAV*(eta + H_west);
        Qx[2][j][i] = (h_east > 0) ? Qx[2][j][i] : -dxfv - 2.0f*GRAV*(eta + H_east);
    }
}


/**
  * Adjusting the slope of L_y, found in Qx[3], to avoid negative values for h on the faces,
  * in the case of dry cells
  */
__device__
void adjustSlopes_y(const int bx, const int by, 
                    float R[3][block_height+4][block_width+4],
                    float Qx[3][block_height+2][block_width+2], // used as if Qx[3][block_height+2][block_width]
                    float Hi[block_height+3][block_width+3],
                    const int& bc_north_, const int& bc_south_,
                    const float* coriolis_f_arr) {
    
    // Need K_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), v (R[2]), H (Hi), g, dx

    
    const int i = threadIdx.x; // values in Qx
    const int k = i + 2; // values in R
    const int H_i = i + 1; // values in Hi
    
    for (int j=threadIdx.y; j<block_height+2; j+=blockDim.y) {
        // i referes to values in Qx
        const int l = j + 1; // values in R
        const int H_j = j; // values in Hi

        // Reconstruct h at east and west faces
        const float eta = R[0][l][k];
        
        float u   = R[1][l][k];
        // Fix south boundary for reconstruction of eta (corresponding to Ly)
        if ((bc_south_ == 1) && (by + l < 2    )) { u = -u; }
        // Fix north boundary for reconstruction of eta (corresponding to Ly)
        if ((bc_north_ == 1) && (by + l > NY+2)) { u = -u; }
        
        // Coriolis in this cell
        const float coriolis_f = coriolisF(coriolis_f_arr, bx+k, by+l, CORIOLIS_F_NX, CORIOLIS_F_NY);

        const float dyfu = DY*coriolis_f*u;
        
        const float H_south = 0.5f*(Hi[H_j  ][H_i] + Hi[H_j  ][H_i+1]);
        const float H_north = 0.5f*(Hi[H_j+1][H_i] + Hi[H_j+1][H_i+1]);
        
        const float h_south = eta + H_south - (Qx[2][j][i] - dyfu)/(2.0f*GRAV);
        const float h_north = eta + H_north + (Qx[2][j][i] - dyfu)/(2.0f*GRAV);
        
        // Adjust if negative water level
        Qx[2][j][i] = (h_south > 0) ? Qx[2][j][i] : dyfu + 2.0f*GRAV*(eta + H_south);
        Qx[2][j][i] = (h_north > 0) ? Qx[2][j][i] : dyfu - 2.0f*GRAV*(eta + H_north);
    }
}




__device__
float3 computeFFaceFlux(const int i, const int j, const int bx,
                float R[3][block_height+4][block_width+4],
                float Qx[3][block_height+2][block_width+2],
                float Hi[block_height+3][block_width+3],
                const float coriolis_fm, const float coriolis_fp, 
                const int& bc_east_, const int& bc_west_,
                const float2 north) {
    const int l = j + 2; //Skip ghost cells (be consistent with reconstruction offsets)
    const int k = i + 1;

    // Skip ghost cells in the Hi buffer
    const int H_i = i+1;
    const int H_j = j+1;
    
    // (u, v) reconstructed at a cell interface from the right (p) and left (m)
    // Variables to reconstruct h from u, v, K, L
    const float eta_bar_p = R[0][l][k+1];
    const float eta_bar_m = R[0][l][k  ];
    const float up = R[1][l][k+1];
    const float um = R[1][l][k  ];
    float vp = R[2][l][k+1];
    float vm = R[2][l][k  ];

    // Check if all dry: if so return zero flux
    if (eta_bar_m == CDKLM_DRY_FLAG && eta_bar_p == CDKLM_DRY_FLAG) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }

    const float2 Rp = make_float2(up - 0.5f*Qx[0][j][i+1], vp - 0.5f*Qx[1][j][i+1]);
    const float2 Rm = make_float2(um + 0.5f*Qx[0][j][i  ], vm + 0.5f*Qx[1][j][i  ]);

    // H is RHx on the given face!
    const float H_face = 0.5f*( Hi[H_j][H_i] + Hi[H_j+1][H_i] );

    // Qx[2] is really dx*Kx
    const float Kx_p = Qx[2][j][i+1];
    const float Kx_m = Qx[2][j][i  ];
    
    // Fix west boundary for reconstruction of eta (corresponding to Kx)
    if ((bc_west_ == 1) && (bx + i + 2 == 2    )) { vm = -vm; }
    // Fix east boundary for reconstruction of eta (corresponding to Kx)
    if ((bc_east_ == 1) && (bx + i + 2 == NX+2)) { vp = -vp; }
    
    //Reconstruct momentum along north
    const float vp_north = up*north.x + vp*north.y;
    const float vm_north = um*north.x + vm*north.y;
    
    // Reconstruct eta
    const float etap = fmaxf(-H_face, eta_bar_p - (Kx_p + DX*coriolis_fp*vp_north)/(2.0f*GRAV));
    const float etam = fmaxf(-H_face, eta_bar_m + (Kx_m + DX*coriolis_fm*vm_north)/(2.0f*GRAV));

    
    // Our flux variables Q=(eta, u, v)
    const float3 Qp = make_float3(etap, Rp.x, Rp.y);
    const float3 Qm = make_float3(etam, Rm.x, Rm.y);

    // Check if wet-dry face: if so balance potential energy of water level
    
    // NOTE: 0-fluxes over wet-dry faces, lead to non-physical waves
    // Hence, we want to control the flux-difference in one way or another
    // (Wall boundary would be desirable, but are inaccessible)
    // The chosen method simply balances the potential-energy flux contribution that acts wet -> dry, by:
    //    F (dry -> wet) = 1/2 g h(wet)^2
    // such that this conserves lakes at rest within numerical precison,
    // but actually ignores wave speeds and simply assumes a symmetric Riemann-fan. 
    // [There are actually now physical values defined on dry cells 
    // and alternative approaches lead to new issues. 
    // One such alternative would be to set "artificial" values on dry cells, e.g.,
    //    Qm = make_float3(hp, 0.0f, 0.0f); 
    // what also conserves lakes at rest but behaves criticial in presence of waves]
    if (eta_bar_m == CDKLM_DRY_FLAG && eta_bar_p != CDKLM_DRY_FLAG) {
        return make_float3(0.0f, 0.5f*GRAV*(Qp.x+H_face)*(Qp.x+H_face), 0.0f);
    }

    if (eta_bar_m != CDKLM_DRY_FLAG && eta_bar_p == CDKLM_DRY_FLAG){
        return make_float3(0.0f, 0.5f*GRAV*(Qm.x+H_face)*(Qm.x+H_face), 0.0f);
    }

    // Computed flux
    return CDKLM16_flux(Qm, Qp, H_face);
}




__device__
float3 computeGFaceFlux(const int i, const int j, const int by,
                float R[3][block_height+4][block_width+4],
                float Qy[3][block_height+2][block_width+2],
                float Hi[block_height+3][block_width+3],
                const float coriolis_fm, const float coriolis_fp, 
                const int& bc_north_, const int& bc_south_,
                const float2 east) {
    const int l = j + 1;
    const int k = i + 2; //Skip ghost cells
    
    // Skip ghost cells in the Hi buffer
    const int H_i = i+1;
    const int H_j = j+1;
    
    // Q at interface from the right and left
    // Variables to reconstruct h from u, v, K, L
    const float eta_bar_p = R[0][l+1][k];
    const float eta_bar_m = R[0][l  ][k];
    float up = R[1][l+1][k];
    float um = R[1][l  ][k];
    const float vp = R[2][l+1][k];
    const float vm = R[2][l  ][k];

    // Check if all dry: if so return zero flux
    if (eta_bar_m == CDKLM_DRY_FLAG && eta_bar_p == CDKLM_DRY_FLAG) {
        return make_float3(0.0f, 0.0f, 0.0f);
    }
    
    const float2 Rp = make_float2(up - 0.5f*Qy[0][j+1][i], vp - 0.5f*Qy[1][j+1][i]);
    const float2 Rm = make_float2(um + 0.5f*Qy[0][j  ][i], vm + 0.5f*Qy[1][j  ][i]);

    // H is RHx on the given face!
    const float H_face = 0.5f*( Hi[H_j][H_i] + Hi[H_j][H_i+1] );

    // Qy[2] is really dy*Ly
    const float Ly_p = Qy[2][j+1][i];
    const float Ly_m = Qy[2][j  ][i];

    // Fix south boundary for reconstruction of eta (corresponding to Ly)
    if ((bc_south_ == 1) && (by + j + 2 == 2    )) { um = -um; }
    // Fix north boundary for reconstruction of eta (corresponding to Ly)
    if ((bc_north_ == 1) && (by + j + 2 == NY+2)) { up = -up; }
    
    // Reconstruct momentum along east
    const float up_east = up*east.x + vp*east.y;
    const float um_east = um*east.x + vm*east.y;
    
    // Reconstruct eta
    const float etap = fmaxf(-H_face, eta_bar_p - ( Ly_p - DY*coriolis_fp*up_east)/(2.0f*GRAV));
    const float etam = fmaxf(-H_face, eta_bar_m + ( Ly_m - DY*coriolis_fm*um_east)/(2.0f*GRAV));

    // Our flux variables Q=(h, v, u)
    // Note that we swap u and v
    const float3 Qp = make_float3(etap, Rp.y, Rp.x);
    const float3 Qm = make_float3(etam, Rm.y, Rm.x);

    // Check if wet-dry face: if so balance potential energy of water level'
    // NOTE: See docu in "computeFFaceFlux"
    if (eta_bar_m == CDKLM_DRY_FLAG && eta_bar_p != CDKLM_DRY_FLAG) {
        return make_float3(0.0f, 0.0f, 0.5f*GRAV*(Qp.x+H_face)*(Qp.x+H_face));
    }

    if (eta_bar_m != CDKLM_DRY_FLAG && eta_bar_p == CDKLM_DRY_FLAG){
        return make_float3(0.0f, 0.0f, 0.5f*GRAV*(Qm.x+H_face)*(Qm.x+H_face));
    }
    
    // Computed flux
    // Note that we swap back u and v
    const float3 flux = CDKLM16_flux(Qm, Qp, H_face);
    return make_float3(flux.x, flux.z, flux.y);
}


__device__ 
void handleWallBC(
                const int& ti_, const int& tj_, 
                const int& tx_, const int& ty_, 
                const int& bc_north_, const int& bc_south_,
                const int& bc_east_, const int& bc_west_,
                float R[3][block_height+4][block_width+4]) {
    const int wall_bc = 1;

    const int i = tx_ + 2; //Skip local ghost cells, i.e., +2
    const int j = ty_ + 2;
        
    if (bc_north_ == wall_bc && tj_ == NY+1) {
        R[0][j+1][i] =  R[0][j][i];
        R[1][j+1][i] =  R[1][j][i];
        R[2][j+1][i] = -R[2][j][i];

        R[0][j+2][i] =  R[0][j-1][i];
        R[1][j+2][i] =  R[1][j-1][i];
        R[2][j+2][i] = -R[2][j-1][i];
    }
    
    if (bc_south_ == wall_bc && tj_ == 2) {
        R[0][j-1][i] =  R[0][j][i];
        R[1][j-1][i] =  R[1][j][i];
        R[2][j-1][i] = -R[2][j][i];

        R[0][j-2][i] =  R[0][j+1][i];
        R[1][j-2][i] =  R[1][j+1][i];
        R[2][j-2][i] = -R[2][j+1][i];
    }
    
    if (bc_east_ == wall_bc && ti_ == NX+1) {
        R[0][j][i+1] =  R[0][j][i];
        R[1][j][i+1] = -R[1][j][i];
        R[2][j][i+1] =  R[2][j][i];

        R[0][j][i+2] =  R[0][j][i-1];
        R[1][j][i+2] = -R[1][j][i-1];
        R[2][j][i+2] =  R[2][j][i-1];
    }
    
    if (bc_west_ == wall_bc && ti_ == 2) {
        R[0][j][i-1] =  R[0][j][i];
        R[1][j][i-1] = -R[1][j][i];
        R[2][j][i-1] =  R[2][j][i];

        R[0][j][i-2] =  R[0][j][i+1];
        R[1][j][i-2] = -R[1][j][i+1];
        R[2][j][i-2] =  R[2][j][i+1];
    }
}



extern "C" {
__global__ void cdklm_swe_2D(
        const float dt_,

        const int step_,    // runge kutta step

        //Input h^n
        float* eta0_ptr_, const int eta0_pitch_,
        float* hu0_ptr_, const int hu0_pitch_,
        float* hv0_ptr_, const int hv0_pitch_,

        //Output h^{n+1}
        float* eta1_ptr_, const int eta1_pitch_,
        float* hu1_ptr_, const int hu1_pitch_,
        float* hv1_ptr_, const int hv1_pitch_,

        //Bathymery
        float* Hi_ptr_, const int Hi_pitch_,
        float* Hm_ptr_, const int Hm_pitch_,
        float land_value_,

        //Coriolis
        const float* coriolis_f_arr,

        //Angle data array
        const float* angle_arr,

        //External forcing parameters
        //Atmospheric pressure
        const float* atmospheric_pressure_current_arr,
        const float* atmospheric_pressure_next_arr,
        const float atmospheric_pressure_t_,
        //Windstress
        const float* wind_stress_x_current_arr,
        const float* wind_stress_x_next_arr,
        const float* wind_stress_y_current_arr,
        const float* wind_stress_y_next_arr,
        const float wind_stress_t_,

        // Boundary conditions (1: wall, 2: periodic, 3: open boundary (flow relaxation scheme))
        // Note: these are packed north, east, south, west boolean bits into an int
        const int boundary_conditions_) {
            
    //const float land_value_ = 1.0e20;


    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of block within domain
    const int bx = blockDim.x * blockIdx.x;
    const int by = blockDim.y * blockIdx.y;

    //Index of cell within domain
    const int ti = blockIdx.x * blockDim.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockIdx.y * blockDim.y + threadIdx.y + 2;

    // Our physical variables
    // Input is [eta, hu, hv]
    // Will store [eta, u, v] (Note u and v are actually computed somewhat down in the code)
    __shared__ float R[3][block_height+4][block_width+4];

    // Our reconstruction variables
    //When computing flux along x-axis, we use
    //Qx = [u_x, v_x, K_x]
    //Then we reuse it as
    //Qx = [u_y, v_y, L_y]
    //to compute the y fluxes
    __shared__ float Qx[3][block_height+2][block_width+2];

    // Bathymetry
    // Need to find H on all faces for the cells in the block (block_height+1, block_width+1)
    // and for one face further out to adjust for the Kx and Ly slope outside of the block
    __shared__ float  Hi[block_height+3][block_width+3];
    
    //Read into shared memory
    // R = [eta, hu, hv]
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by + j, 0, NY+3); // Out of bounds

        //Compute the pointer to current row in the arrays
        float* const eta_row = (float*) ((char*) eta0_ptr_ + eta0_pitch_*l);
        float* const hu_row = (float*) ((char*) hu0_ptr_ + hu0_pitch_*l);
        float* const hv_row = (float*) ((char*) hv0_ptr_ + hv0_pitch_*l);

        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx + i, 0, NX+3); // Out of bounds

            R[0][j][i] = eta_row[k];
            R[1][j][i] = hu_row[k];
            R[2][j][i] = hv_row[k];
        }
    }
    __syncthreads();
    

    // Read Hi into shared memory
    // Read intersections on all non-ghost cells
    for(int j=ty; j < block_height+3; j+=blockDim.y) {
        // Skip ghost cells and
        const int l = clamp(by+j+1, 1, NY+4);
        float* const Hi_row = (float*) ((char*) Hi_ptr_ + Hi_pitch_*l);
        for(int i=tx; i < block_width+3; i+=blockDim.x) {
            const int k = clamp(bx+i+1, 1, NX+4);

            Hi[j][i] = Hi_row[k];
            
            if (fabsf(Hi[j][i] - land_value_) < CDKLM_DRY_EPS) {
                Hi[j][i] = CDKLM_DRY_FLAG;
            }
        }
    }
    __syncthreads();
    

    //Fix boundary conditions
    //This must match code in CDKLM16.py:callKernel(...)
    const int bc_north = (boundary_conditions_ >> 24) & 0xFF;
    const int bc_south = (boundary_conditions_ >> 16) & 0xFF;
    const int bc_east = (boundary_conditions_ >> 8) & 0xFF;
    const int bc_west = (boundary_conditions_ >> 0) & 0xFF;
    
    if (boundary_conditions_ > 0) {
        // These boundary conditions are dealt with inside shared memory
        handleWallBC(ti, tj,
                tx, ty,
                bc_north, bc_south,
                bc_east, bc_west,
                R);
    }

    __syncthreads();
    
    // Compensate for one layer of ghost cells
    float Hm = 0.25f*(Hi[ty+1][tx+1] + Hi[ty+2][tx+1] + Hi[ty+1][tx+2] + Hi[ty+2][tx+2]);

    //Create our "steady state" reconstruction variables (u, v)
    // K and L are never stored, but computed where needed.
    // R = [eta, hu, hv] --> R = [eta, u, v]
    for (int j=ty; j<block_height+4; j+=blockDim.y) {
        const int l = clamp(by+j, 0, NY+3);
        float* const Hm_row = (float*) ((char*) Hm_ptr_ + Hm_pitch_*l);
        for (int i=tx; i<block_width+4; i+=blockDim.x) {
            const int k = clamp(bx+i, 0, NX+3);

            // h = eta + H
            const float local_Hm = Hm_row[k];
            //const float local_Hm = 0.25f*(Hi[j][i] + Hi[j+1][i] + Hi[j][i+1] + Hi[j+1][i+1]);
            const float h = R[0][j][i] + local_Hm;
            
            //Check if this cell is actually dry (or land)
            //NOTE: This requires that all four corners of a cell are dry to be considered dry cell
            if (fabsf(local_Hm - land_value_) <= CDKLM_DRY_EPS) {
                R[0][j][i] = CDKLM_DRY_FLAG;
                R[1][j][i] = 0.0f;
                R[2][j][i] = 0.0f;
            }
            // Check if the cell is almost dry
            else if (h < KPSIMULATOR_DESING_EPS) {
                
                if (h <= KPSIMULATOR_DEPTH_CUTOFF) {
                    R[0][j][i] = -local_Hm + KPSIMULATOR_DEPTH_CUTOFF;
                    R[1][j][i] = 0.0f;
                    R[2][j][i] = 0.0f;
                }
                else {                
                    // Desingularizing u and v
                    //R[0][j][i] = h - local_Hm;
                    R[1][j][i] = desingularize(h, R[1][j][i], KPSIMULATOR_DESING_EPS); 
                    R[2][j][i] = desingularize(h, R[2][j][i], KPSIMULATOR_DESING_EPS); 
                }
            }
            // Wet cells
            else {
                R[1][j][i] /= h;
                R[2][j][i] /= h;
            }

            
        }
    }
    __syncthreads();

    // Store desingulized hu and hv
    //Skip local ghost cells, i.e., +2
    float hu = 0.0f;
    float hv = 0.0f;
    if ((R[0][ty + 2][tx + 2] + Hm) > KPSIMULATOR_DEPTH_CUTOFF) {
        hu = R[1][ty + 2][tx + 2]*(R[0][ty + 2][tx + 2] + Hm);
        hv = R[2][ty + 2][tx + 2]*(R[0][ty + 2][tx + 2] + Hm);
    }




    //Reconstruct slopes along x axis
    // Write result into shmem Qx = [u_x, v_x, K_x]*dx
    // Qx is used as if its size was Qx[3][block_height][block_width + 2]
    for (int j=ty; j<block_height; j+=blockDim.y) {
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<block_width+2; i+=blockDim.x) {
            const int k = i + 1;

            const float left_eta   = R[0][l][k-1];
            const float center_eta = R[0][l][k  ];
            const float right_eta  = R[0][l][k+1];

            const float left_u   = R[1][l][k-1];
            const float center_u = R[1][l][k  ];
            const float right_u  = R[1][l][k+1];
        
            float left_v   = R[2][l][k-1];
            float center_v = R[2][l][k  ];
            float right_v  = R[2][l][k+1];
            
            Qx[0][j][i] = minmodSlope(left_u, center_u, right_u, THETA);
            Qx[1][j][i] = minmodSlope(left_v, center_v, right_v, THETA);
            
            // Enforce wall boundary conditions for Kx:
            int global_thread_id_x = bx + i + 1; // index including ghost cells'
            // Western BC
            if (bc_west == 1) {
                if (global_thread_id_x < 3    ) { left_v   = -left_v;   }
                if (global_thread_id_x < 2    ) { center_v = -center_v; }
            }
            // Eastern BC
            if (bc_east == 1) {
                if (global_thread_id_x > NX  ) { right_v  = -right_v;  }
                if (global_thread_id_x > NX+1) { center_v = -center_v; }
            }
            
            // Get north vector for thread (bx + k, by +l)
            const float2 local_north = getNorth(angle_arr, bx+k, by+l, ANGLE_NX, ANGLE_NY);
            
            const float left_coriolis_f   = coriolisF(coriolis_f_arr, bx+k-1, by+l, CORIOLIS_F_NX, CORIOLIS_F_NY);
            const float center_coriolis_f = coriolisF(coriolis_f_arr, bx+k  , by+l, CORIOLIS_F_NX, CORIOLIS_F_NY);
            const float right_coriolis_f  = coriolisF(coriolis_f_arr, bx+k+1, by+l, CORIOLIS_F_NX, CORIOLIS_F_NY);
            
            const float left_fv  = (local_north.x*left_u + local_north.y*left_v)*left_coriolis_f;
            const float center_fv = (local_north.x*center_u + local_north.y*center_v)*center_coriolis_f;
            const float right_fv  = (local_north.x*right_u + local_north.y*right_v)*right_coriolis_f;
            
            const float V_constant = DX/(2.0f*GRAV);

            // Qx[2] = Kx, which we need to find differently than ux and vx
            const float backward = THETA*GRAV*(center_eta - left_eta   - V_constant*(center_fv + left_fv ) );
            const float central  =  0.5f*GRAV*(right_eta  - left_eta   - V_constant*(right_fv + 2.0f*center_fv + left_fv) );
            const float forward  = THETA*GRAV*(right_eta  - center_eta - V_constant*(center_fv + right_fv) );

            // Qx[2] is really dx*Kx
            Qx[2][j][i] = minmodRaw(backward, central, forward);

        }
    }
    __syncthreads();
        
    // Adjust K_x slopes to avoid negative h = eta + H
    // Need K_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), v (R[2]), H (Hi), g, dx
    adjustSlopes_x(bx, by,
                   R, Qx, Hi,
                   bc_east, bc_west,
                   coriolis_f_arr);
    __syncthreads();
   
    float3 flux_diff;
    
    // Get Coriolis terms needed for fluxes etc.
    const float coriolis_f_central = coriolisF(coriolis_f_arr, ti, tj, CORIOLIS_F_NX, CORIOLIS_F_NY);
    // North and east vector in xy-coordinate system
    const float2 north = getNorth(angle_arr, ti, tj, ANGLE_NX, ANGLE_NY);
    const float2 east = make_float2(north.y, -north.x);
    
    { //Scope
        const float coriolis_f_left    = coriolisF(coriolis_f_arr, ti-1, tj, CORIOLIS_F_NX, CORIOLIS_F_NY);
        const float coriolis_f_right   = coriolisF(coriolis_f_arr, ti+1, tj, CORIOLIS_F_NX, CORIOLIS_F_NY);

        // Compute flux along x axis
        flux_diff = (  
                computeFFaceFlux(
                    tx+1, ty, bx, 
                    R, Qx, Hi,
                    coriolis_f_central, coriolis_f_right, 
                    bc_east, bc_west,
                    north)
                - 
                computeFFaceFlux(
                    tx , ty, bx,  
                    R, Qx, Hi,
                    coriolis_f_left, coriolis_f_central, 
                    bc_east, bc_west, 
                    north)) / DX;
    }
    __syncthreads();
    
    // Reconstruct eta_west, eta_east for use in bathymetry source term
    const float eta_west = R[0][ty+2][tx+2] - (Qx[2][ty][tx+1] + DX*coriolis_f_central*R[2][ty+2][tx+2])/(2.0f*GRAV);
    const float eta_east = R[0][ty+2][tx+2] + (Qx[2][ty][tx+1] + DX*coriolis_f_central*R[2][ty+2][tx+2])/(2.0f*GRAV);
    
    __syncthreads();
    
    //Reconstruct slopes along y axis
    // Write result into shmem Qx = [u_y, v_y, L_y]*dy
    // Qx is now used as if its size was Qx[3][block_height+2][block_width]

    for (int j=ty; j<block_height+2; j+=blockDim.y) {
        const int l = j + 1;
        for (int i=tx; i<block_width; i+=blockDim.x) {
            const int k = i + 2; //Skip ghost cells
            // Qy[2] = Ly, which we need to find differently than uy and vy
            const float lower_eta  = R[0][l-1][k];
            const float center_eta = R[0][l  ][k];
            const float upper_eta  = R[0][l+1][k];

            float lower_u  = R[1][l-1][k];
            float center_u = R[1][l  ][k];
            float upper_u  = R[1][l+1][k];

            const float lower_v  = R[2][l-1][k];
            const float center_v = R[2][l  ][k];
            const float upper_v  = R[2][l+1][k];
            
            Qx[0][j][i] = minmodSlope(lower_u, center_u, upper_u, THETA);
            Qx[1][j][i] = minmodSlope(lower_v, center_v, upper_v, THETA);

            // Enforce wall boundary conditions for Ly
            int global_thread_id_y = by + j + 1; // index including ghost cells
            // southern BC
            if (bc_south == 1) {
                if (global_thread_id_y < 3    ) { lower_u  = -lower_u;  }
                if (global_thread_id_y < 2    ) { center_u = -center_u; }
            }
            // northern BC
            if (bc_north == 1) {
                if (global_thread_id_y > NY  ) { upper_u  = -upper_u;  }
                if (global_thread_id_y > NY+1) { center_u = -center_u; }
            }
            
            // Get north and east vectors for thread (bx + k, by +l)
            const float2 local_north = getNorth(angle_arr, bx+k, by+l, ANGLE_NX, ANGLE_NY);
            const float2 local_east = make_float2(local_north.y, -local_north.x);
            
            const float lower_coriolis_f  = coriolisF(coriolis_f_arr, bx+k, by+l-1, CORIOLIS_F_NX, CORIOLIS_F_NY);
            const float center_coriolis_f = coriolisF(coriolis_f_arr, bx+k, by+l  , CORIOLIS_F_NX, CORIOLIS_F_NY);
            const float upper_coriolis_f  = coriolisF(coriolis_f_arr, bx+k, by+l+1, CORIOLIS_F_NX, CORIOLIS_F_NY);

            const float lower_fu  = (local_east.x*lower_u  + local_east.y*lower_v )*lower_coriolis_f;
            const float center_fu = (local_east.x*center_u + local_east.y*center_v)*center_coriolis_f;
            const float upper_fu  = (local_east.x*upper_u  + local_east.y*upper_v )*upper_coriolis_f;

            const float U_constant = DY/(2.0f*GRAV);

            const float backward = THETA*GRAV*(center_eta - lower_eta  + U_constant*(center_fu + lower_fu ) );
            const float central  =  0.5f*GRAV*(upper_eta  - lower_eta  + U_constant*(upper_fu + 2.0f*center_fu + lower_fu) );
            const float forward  = THETA*GRAV*(upper_eta  - center_eta + U_constant*(center_fu + upper_fu) );

            // Qy[2] is really dy*Ly
            Qx[2][j][i] = minmodRaw(backward, central, forward);
        }
    }
    __syncthreads();

    // Adjust L_y slopes to avoid negative h = eta + H
    // Need L_x (Qx[2]), coriolis parameter (f, beta), eta (R[0]), u (R[1]), H (Hi), g, dx
    adjustSlopes_y(bx, by,
                   R, Qx, Hi,
                   bc_north, bc_south,
                   coriolis_f_arr);
    __syncthreads();
    
    
    if (! ONE_DIMENSIONAL)
    { // scope
        const float coriolis_f_lower   = coriolisF(coriolis_f_arr, ti, tj-1, CORIOLIS_F_NX, CORIOLIS_F_NY);
        const float coriolis_f_upper   = coriolisF(coriolis_f_arr, ti, tj+1, CORIOLIS_F_NX, CORIOLIS_F_NY);
    
        //Compute fluxes along the y axis
        flux_diff = flux_diff + 
            (computeGFaceFlux(
                tx, ty+1, by, 
                R, Qx, Hi, 
                coriolis_f_central, coriolis_f_upper, 
                bc_north, bc_south, 
                east)
            - 
            computeGFaceFlux(
                tx, ty, by,  
                R, Qx, Hi, 
                coriolis_f_lower, coriolis_f_central, 
                bc_north, bc_south, 
                east)) / DY;
        __syncthreads();
    }

    // Reconstruct eta_north, eta_south for use in bathymetry source term
    const float eta_south = R[0][ty+2][tx+2] - (Qx[2][ty+1][tx] - DY*coriolis_f_central*R[1][ty+2][tx+2])/(2.0f*GRAV);
    const float eta_north = R[0][ty+2][tx+2] + (Qx[2][ty+1][tx] - DY*coriolis_f_central*R[1][ty+2][tx+2])/(2.0f*GRAV);
    __syncthreads();
    
    //Sum fluxes and advance in time for all internal cells
    if (ti > 1 && ti < NX+2 && tj > 1 && tj < NY+2) {
        //Skip local ghost cells, i.e., +2
        const int i = tx + 2; 
        const int j = ty + 2;
        
        // Skip local ghost cells for Hi
        const int H_i = tx + 1;
        const int H_j = ty + 1;

        // Source terms (wind, coriolis, bathymetry)
        float st1 = 0.0f;
        float st2 = 0.0f;
        
        const float h = R[0][j][i] + Hm;
        //If wet cell
        if (h >= KPSIMULATOR_DEPTH_CUTOFF) {
            // If not land
            if (R[0][j][i] != CDKLM_DRY_FLAG) {
                // Bottom topography source terms!
                // -g*(eta + H)*(-1)*dH/dx   * dx
                const float RHxp = 0.5f*( Hi[H_j  ][H_i+1] + Hi[H_j+1][H_i+1] );
                const float RHxm = 0.5f*( Hi[H_j  ][H_i  ] + Hi[H_j+1][H_i  ] );
                const float RHyp = 0.5f*( Hi[H_j+1][H_i  ] + Hi[H_j+1][H_i+1] );
                const float RHym = 0.5f*( Hi[H_j  ][H_i  ] + Hi[H_j  ][H_i+1] );
                
                
                //Project momenta onto north/east axes
                const float hu_east =  hu*east.x + hv*east.y;
                const float hv_north = hu*north.x + hv*north.y;
                
                //Convert momentums between east/north due to Coriolis
                const float hu_east_cor = coriolis_f_central*hv_north;
                const float hv_north_cor = -coriolis_f_central*hu_east;

                //Project back to x/y-coordinate system
                const float2 up = make_float2(-north.x, north.y);
                const float2 right = make_float2(up.y, -up.x);
                const float hu_cor = right.x*hu_east_cor + right.y*hv_north_cor;
                const float hv_cor = up.x*hu_east_cor + up.y*hv_north_cor;

                // Atmospheric pressure
#if USE_DIRECT_LOOKUP
                const float2 atm_p_central_diff = atmospheric_pressure_central_diff_lookup(atmospheric_pressure_current_arr, atmospheric_pressure_next_arr, atmospheric_pressure_t_,  ti, tj, ATMOS_PRES_NX);
#else
                const float2 atm_p_central_diff = atmospheric_pressure_central_diff(atmospheric_pressure_current_arr, atmospheric_pressure_next_arr, atmospheric_pressure_t_,  ti+0.5, tj+0.5, NX+4, NY+4, ATMOS_PRES_NX, ATMOS_PRES_NY);
#endif
                // TODO: We might want to use the mean of the reconstructed eta's at the faces here, instead of R[0]...
                //const float bathymetry1 = GRAV*(R[0][j][i] + Hm)*H_x;
                //const float bathymetry2 = GRAV*(R[0][j][i] + Hm)*H_y;

            {
                // Atmospheric pressure
                const float atm_pressure_x = -atm_p_central_diff.x*h/(2.0f*DX*RHO_O);
                // Wind
                const float X = WIND_STRESS_FACTOR * windStress(wind_stress_x_current_arr, wind_stress_x_next_arr, wind_stress_t_, ti+0.5, tj+0.5, NX+4, NY+4, WIND_STRESS_X_NX, WIND_STRESS_X_NY);
                // Coriolis
                const float eta_we = 0.5f*(eta_west  + eta_east);
                // Bottom topography
                const float H_x = RHxp - RHxm;
                const float bathymetry1 = GRAV*(eta_we + Hm)*H_x;
                // Total source terms
                st1 = X + hu_cor + atm_pressure_x + bathymetry1/DX;
            }
            {   
                // Atmospheric pressure
                const float atm_pressure_y = -atm_p_central_diff.y*h/(2.0f*DY*RHO_O);
                // Wind
                const float Y = WIND_STRESS_FACTOR * windStress(wind_stress_y_current_arr, wind_stress_y_next_arr, wind_stress_t_, ti+0.5, tj+0.5, NX+4, NY+4, WIND_STRESS_Y_NX, WIND_STRESS_Y_NY);
                // Coriolis
                const float eta_sn = 0.5f*(eta_north + eta_south);
                const float H_y = RHyp - RHym;
                // Bottom topography
                const float bathymetry2 = GRAV*(eta_sn + Hm)*H_y;
                // Total source terms
                st2 = Y + hv_cor + atm_pressure_y + bathymetry2/DY;
            }
            }
        }

        
        const float L1  = - flux_diff.x;
        const float L2  = - flux_diff.y + st1;
        const float L3  = - flux_diff.z + st2;

        float* const eta_row = (float*) ((char*) eta1_ptr_ + eta1_pitch_*tj);
        float* const hu_row  = (float*) ((char*) hu1_ptr_  +  hu1_pitch_*tj);
        float* const hv_row  = (float*) ((char*) hv1_ptr_  +  hv1_pitch_*tj);

        float updated_eta;
        float updated_hu;
        float updated_hv;
        
        if (RK_ORDER < 3) {

#ifdef use_linear_friction
            const float C = 2.0f*FRIC*dt_/(R[0][j][i] + Hm);
#else
            float C = 0.0;
            if (FRIC > 0.0) {
                if (h < KPSIMULATOR_DESING_EPS) {
                    const float u = desingularize(h, hu, KPSIMULATOR_DESING_EPS);
                    const float v = desingularize(h, hv, KPSIMULATOR_DESING_EPS);
                    C = dt_*FRIC*desingularize(h, sqrt(u*u+v*v), KPSIMULATOR_DESING_EPS);
                }
                else {
                    const float u = hu/h;
                    const float v = hv/h;
                    C = dt_*FRIC*sqrt(u*u+v*v)/h;
                }
            }
#endif
            
            if  (step_ == 0) {
                //First step of RK2 ODE integrator

                updated_eta =  R[0][j][i] + dt_*L1;
                updated_hu  = (hu + dt_*L2) / (1.0f + C);
                updated_hv  = (hv + dt_*L3) / (1.0f + C);
            }
            else if (step_ == 1) {
                //Second step of RK2 ODE integrator

                //First read Q^n
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                //Compute Q^n+1
                const float eta_b = 0.5f*(eta_a + (R[0][j][i] + dt_*L1));
                const float hu_b  = 0.5f*( hu_a + (hu + dt_*L2));
                const float hv_b  = 0.5f*( hv_a + (hv + dt_*L3));


                //Write to main memory
                updated_eta = eta_b;
                updated_hu  =  hu_b / (1.0f + 0.5f*C);
                updated_hv  =  hv_b / (1.0f + 0.5f*C);

            }
        }


        else if (RK_ORDER == 3) {
            // Third order Runge Kutta - only valid if r_ = 0.0 (no friction)

            if (step_ == 0) {
                //First step of RK3 ODE integrator
                // q^(1) = q^n + dt*L(q^n)

                updated_eta =  R[0][j][i] + dt_*L1;
                updated_hu  = (hu + dt_*L2);
                updated_hv  = (hv + dt_*L3);

            } else if (step_ == 1) {
                // Second step of RK3 ODE integrator
                // Q^(2) = 3/4 Q^n + 1/4 ( Q^(1) + dt*L(Q^(1)) )
                // Q^n is here in h1, but will be used in next iteration as well --> write to h0

                // First read Q^n:
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                // Compute Q^(2):
                const float eta_b = 0.75f*eta_a + 0.25f*(R[0][j][i] + dt_*L1);
                const float hu_b  = 0.75f* hu_a + 0.25f*(hu + dt_*L2);
                const float hv_b  = 0.75f* hv_a + 0.25f*(hv + dt_*L3);

                // Write output to the input buffer:
                updated_eta = eta_b;
                updated_hu  =  hu_b;
                updated_hv  =  hv_b;

            } else if (step_ == 2) {
                // Third step of RK3 ODE integrator
                // Q^n+1 = 1/3 Q^n + 2/3 (Q^(2) + dt*L(Q^(2))

                // First read Q^n:
                const float eta_a = eta_row[ti];
                const float hu_a  =  hu_row[ti];
                const float hv_a  =  hv_row[ti];

                // Compute Q^n+1:
                const float eta_b = (eta_a + 2.0f*(R[0][j][i] + dt_*L1)) / 3.0f;
                const float hu_b  = ( hu_a + 2.0f*(hu + dt_*L2)) / 3.0f;
                const float hv_b  = ( hv_a + 2.0f*(hv + dt_*L3)) / 3.0f;

                //Write to main memory
                updated_eta = eta_b;
                updated_hu  =  hu_b;
                updated_hv  =  hv_b;
            }
        }
    

        const float updated_h = updated_eta + Hm;
        if ((updated_h <= KPSIMULATOR_DEPTH_CUTOFF) ) { 
            updated_eta = -Hm + KPSIMULATOR_DEPTH_CUTOFF;
            updated_hu  = 0.0f;
            updated_hv  = 0.0f;
        }

        if ( (RK_ORDER == 3) && (step_ == 1) ) {
            float* const eta_out_row = (float*) ((char*) eta0_ptr_ + eta0_pitch_*tj);
            float* const hu_out_row  = (float*) ((char*)  hu0_ptr_ +  hu0_pitch_*tj);
            float* const hv_out_row  = (float*) ((char*)  hv0_ptr_ +  hv0_pitch_*tj);

            eta_out_row[ti] = fmaxf(-Hm + KPSIMULATOR_DEPTH_CUTOFF, updated_eta);
            hu_out_row[ti]  = updated_hu;
            hv_out_row[ti]  = updated_hv;
        } else {
            eta_row[ti] = fmaxf(-Hm + KPSIMULATOR_DEPTH_CUTOFF, updated_eta);
            hu_row[ti]  = updated_hu;
            hv_row[ti]  = updated_hv;
        }
    }
}

}
